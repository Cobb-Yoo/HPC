
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000
#define THREADS 100

__global__ void saxpy(float *A, float*B, float X, float Y){
   int i = blockDim.x*blockIdx.x+threadIdx.x;

   B[i] = A[i]*X;
   B[i] += Y;
}

int main()
{
   float A[N], B[N], B2[N], X, Y;
   float *A_d, *B_d;
   int i;

   dim3 dimBlock(THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);

   for (i=0; i<N; i++)
      A[i] = i*2;

   X = 1.23;
   Y = 2.34;
   for (i=0; i<N; i++)
      B2[i] = A[i]*X + Y; // B2 is used for checking

   hipMalloc((void**) &A_d, sizeof(float)*N);
   hipMalloc((void**) &B_d, sizeof(float)*N);

   hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);
   saxpy<<<dimGrid, dimBlock>>>(A_d, B_d, X, Y);

   hipMemcpy(B, B_d, sizeof(float)*N, hipMemcpyDeviceToHost);

   for (i=0; i<N; i++)
      if (fabs(B[i]-B2[i]) > 0.001)
         printf("%d: %f %f\n",i, B[i], B2[i]);

   hipFree(A_d);
   hipFree(B_d);
}
